#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/extrema.h"
#include "thrust/reduce.h"
#include "thrust/functional.h"
#include "thrust/execution_policy.h"

#include "gdal_util.h"
#include "cpl_conv.h"


template<typename T1, typename T2>
struct type2_type
{
	__host__ __device__ T2 operator()(const T1& x) const
	{
		return static_cast<T2>(x);
	}
};

struct variance : std::unary_function<us, double>
{
	variance(double m) : mean(m) { }
	const double mean;
	__host__ __device__ double operator()(us data) const
	{
		return std::pow(data - mean, 2.0);
	}
};

// atomicCAS 暂不支持 uc

__global__ void pixels_std(us* data, uc* res, const ull size, us band_max, us band_min, us uc_max, us uc_min, float k, float b)
{
	ui tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= size) return;

	const us d = data[tid];
	us v;
	if (d == band_min)
		v = band_min;
	else if (d <= uc_min)
		v = band_min;
	else if (d >= uc_max)
		v = band_max;
	else if (k * d + b < band_min)
		v = band_min;
	else if (k * d + b > band_max)
		v = band_max;
	else if (k * d + b > band_min && k * d + b < band_max)
		v = k * d + b;
	else
		v = d;

	res[tid] = static_cast<uc>(v);
}

int main(int argc, char* argv[])
{
	// 16bit 转 8bit
	GDALAllRegister();

	char psz_filename[1024] = "D:\\统筹影像\\cuda\\PAN31.TIF";
	char psz_filename_new[1024] = "D:\\统筹影像\\cuda\\PANNew.TIF";

	GDALDriver* tifDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	const GDALDatasetH dataset_16bit = GDALOpen(psz_filename, GA_Update);
	
	// if (dataset == NULL)
	raster_info ri;
	get_raster_info(dataset_16bit, &ri);

	GDALDataset* dataset_8bit = tifDriver->Create(psz_filename_new, ri.width, ri.height, GDALGetRasterCount(dataset_16bit), GDT_Byte,NULL);
	dataset_8bit->SetGeoTransform(ri.geo_transform);
	dataset_8bit->SetProjection(ri.projection);

	printf("Size is %dx%dx%d\n",
		ri.width,
		ri.height,
		GDALGetRasterCount(dataset_8bit));
	printf("Pixel Size = (%.6f,%.6f)\n",
		ri.geo_transform[1], ri.geo_transform[5]);

	hipError_t status;
	GDALRasterBandH h_band_16;
	GDALRasterBandH h_band_8;
	const int x_size = ri.width;
	const int y_size = ri.height;
	const ull size = x_size * y_size;
	const ull malloc_size = sizeof(us) * x_size * y_size;

	// 原影像
	us* h_data;
	uc* h_res;
	h_data = (us*)CPLMalloc(malloc_size);
	h_res = (uc*)CPLMalloc(size);
	// 新影像
	us* d_data;
	uc* d_res;
	status = hipMalloc((void**)&d_data, malloc_size);
	status = hipMalloc((void**)&d_res, size);
	for (int i = 0; i < 3; ++i)
	{
		h_band_16 = GDALGetRasterBand(dataset_16bit, i + 1);
		h_band_8 = GDALGetRasterBand(dataset_8bit, i + 1);
		GDALRasterIO(h_band_16, GF_Read, 0, 0, x_size, y_size,
			h_data, x_size, y_size, GDT_UInt16, 0, 0);
		// 拷贝数组大小有误
		status = hipMemcpy(d_data, h_data, malloc_size, hipMemcpyHostToDevice);
		thrust::device_ptr<us> ptr(d_data);
		// 数组越界时抛出 msg:extrema failed to synchronize
		// 最大值最小值仅为测试函数
		const auto max_iter = thrust::max_element(ptr, ptr + size);
		const auto min_iter = thrust::min_element(ptr, ptr + size);
		us band_max = *max_iter;
		us band_min = *min_iter;
		band_max = 255;
		band_min = 0;
		// cpu 执行
		// auto band_sum_cpu = thrust::reduce(thrust::host, h_data, h_data + size, (ull)0);
		// gpu 执行
		auto band_sum = thrust::reduce(ptr, ptr + size, (ull)0);
		double band_mean = band_sum / (double)size;
		// 方差 (val-mean)*(val-mean)
		auto band_std2 = thrust::transform_reduce(ptr, ptr + size, variance(band_mean), (double)0, thrust::plus<double>());

		double band_std = std::sqrt(band_std2/(double)(size-1));
		float kn = 2.5;
		float uc_max = band_mean + kn * band_std;
		float uc_min = band_mean - kn * band_std;
		float k = (band_max - band_min) / (uc_max - uc_min);
		float b = (uc_max * band_min - uc_min * band_max) / (uc_max - uc_min);
		if (uc_min <= 0)
			uc_min = 0;

		// 标准差
		const ui block_size = 128;
		const ui grid_size = (size - 1) / block_size + 1;
		pixels_std << <grid_size, block_size >> > (d_data, d_res, size, band_max, band_min, uc_max, uc_min, k, b);

		hipDeviceSynchronize();

		hipMemcpy(h_res, d_res, size, hipMemcpyDeviceToHost);
		//
		GDALRasterIO(h_band_8, GF_Write, 0, 0, x_size, y_size,
			h_res, x_size, y_size, GDT_Byte,0, 0);
	}
	hipFree(d_data);
	hipFree(d_res);
	CPLFree(h_data);
	CPLFree(h_res);
	
	GDALClose(dataset_16bit);
	GDALClose(dataset_8bit);
	
	return 0;
}

